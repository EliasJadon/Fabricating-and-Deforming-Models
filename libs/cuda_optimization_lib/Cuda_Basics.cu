#include "hip/hip_runtime.h"
#include "Cuda_Basics.cuh"


namespace Cuda {
	hinge newHinge(int f0, int f1) {
		hinge a;
		a.f0 = f0;
		a.f1 = f1;
		return a;
	}

	void CheckErr(const hipError_t cudaStatus, const int ID) {
		if (cudaStatus != hipSuccess) {
			std::cout << "Error!!!" << std::endl;
			std::cout << "ID = " << ID << std::endl;
			std::cout << "cudaStatus:\t" << hipGetErrorString(cudaStatus) << std::endl;
			std::cout << "Last Error:\t" << hipGetErrorString(hipGetLastError()) << std::endl;
			exit(1);
		}
	}
	
	void initCuda() {
		view_device_properties();
		// Choose which GPU to run on, change this on a multi-GPU system.
		CheckErr(hipSetDevice(0));
		std::cout << "hipSetDevice successfully!\n";
		
	}
	void StopCudaDevice() {
		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		CheckErr(hipDeviceReset());
		std::cout << "hipDeviceReset successfully!\n";
	}
	void view_device_properties() {
		int nDevices;
		hipGetDeviceCount(&nDevices);
		for (int i = 0; i < nDevices; i++) {
			hipDeviceProp_t prop;
			CheckErr(hipGetDeviceProperties(&prop, i));
			std::cout << "Device Number: " << i << std::endl;
			std::cout << "\tName: " << prop.name << std::endl;
			std::cout << "\tMemory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
			std::cout << "\tMemory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
			std::cout << "\tPeak Memory Bandwidth (GB/s): " << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6 << std::endl;
			std::cout << "\tprop.maxThreadsPerBlock = " << prop.maxThreadsPerBlock << std::endl;
			std::cout << "\tprop.maxThreadsDim[0] = " << prop.maxThreadsDim[0] << std::endl;
			std::cout << "\tprop.maxThreadsDim[1] = " << prop.maxThreadsDim[1] << std::endl;
			std::cout << "\tprop.maxThreadsDim[2] = " << prop.maxThreadsDim[2] << std::endl;
			std::cout << "\tprop.maxGridSize[0] = " << prop.maxGridSize[0] << std::endl;
			std::cout << "\tprop.maxGridSize[1] = " << prop.maxGridSize[1] << std::endl;
			std::cout << "\tprop.maxGridSize[2] = " << prop.maxGridSize[2] << std::endl;
		}
	}

	__global__ void copyArraysKernel(double* a, const double* b) {
		int index = blockIdx.x;
		a[index] = b[index];
	}

	void copyArrays(Array<double>& a, const Array<double>& b) {
		copyArraysKernel << <a.size, 1 >> > (a.cuda_arr, b.cuda_arr);
		CheckErr(hipDeviceSynchronize());
	}
}

