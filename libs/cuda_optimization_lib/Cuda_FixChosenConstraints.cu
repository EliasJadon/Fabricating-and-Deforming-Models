#include "hip/hip_runtime.h"
#include "Cuda_FixChosenConstraints.cuh"
#include "Cuda_Minimizer.cuh"

namespace Utils_Cuda_FixChosenConstraints {
	__device__ double3 sub(const double3 a, const double3 b)
	{
		return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
	}
	__device__ double dot(const double3 a, const double3 b)
	{
		return a.x * b.x + a.y * b.y + a.z * b.z;
	}
	__device__ double squared_norm(const double3 a)
	{
		return dot(a, a);
	}
	template<typename T>
	__global__ void setZeroKernel(T* vec)
	{
		vec[blockIdx.x] = 0;
	}
	template <unsigned int blockSize, typename T>
	__device__ void warpReduce(volatile T* sdata, unsigned int tid) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	__device__ double atomicAdd(double* address, double val, int flag)
	{
		unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
		unsigned long long int old = *address_as_ull, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));
			// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
		} while (assumed != old);
		return __longlong_as_double(old);
	}

	template<unsigned int blockSize>
	__global__ void EnergyKernel(
		double* resAtomic,
		const double* curr_x,
		const unsigned int size,
		const int* Const_Ind,
		const double3* Const_Pos,
		const unsigned int startX,
		const unsigned int startY,
		const unsigned int startZ)
	{
		//init data
		extern __shared__ double energy_value[blockSize];
		unsigned int tid = threadIdx.x;
		unsigned int Global_idx = blockIdx.x * blockSize + tid;
		energy_value[tid] = 0;
		
		if (Global_idx < size) {
			double3 Vi = make_double3(
				curr_x[Const_Ind[Global_idx] + startX],
				curr_x[Const_Ind[Global_idx] + startY],
				curr_x[Const_Ind[Global_idx] + startZ]
			);
			energy_value[tid] = squared_norm(sub(Vi, Const_Pos[Global_idx]));
		}

		__syncthreads();

		if (blockSize >= 1024) { if (tid < 512) { energy_value[tid] += energy_value[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { energy_value[tid] += energy_value[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { energy_value[tid] += energy_value[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { energy_value[tid] += energy_value[tid + 64]; } __syncthreads(); }
		if (tid < 32) warpReduce<blockSize, double>(energy_value, tid);
		if (tid == 0) atomicAdd(resAtomic, energy_value[0], 0);
	}

	__global__ void gradientKernel(
		double* grad,
		const double* X,
		const unsigned int startX,
		const unsigned int startY,
		const unsigned int startZ,
		const int* Const_Ind,
		const double3* Const_Pos,
		const unsigned int size)
	{
		int i = blockIdx.x;
		if (i < size) {
			if (threadIdx.x == 0)
				grad[Const_Ind[i] + startX] = 2 * (X[Const_Ind[i] + startX] - Const_Pos[i].x);
			if (threadIdx.x == 1)
				grad[Const_Ind[i] + startY] = 2 * (X[Const_Ind[i] + startY] - Const_Pos[i].y);
			if (threadIdx.x == 2)
				grad[Const_Ind[i] + startZ] = 2 * (X[Const_Ind[i] + startZ] - Const_Pos[i].z);
		}
	}
}


void Cuda_FixChosenConstraints::value(Cuda::Array<double>& curr_x) {
	Utils_Cuda_FixChosenConstraints::setZeroKernel << <1, 1>> > (EnergyAtomic.cuda_arr);
	const unsigned int s = Const_Ind.size;
	Utils_Cuda_FixChosenConstraints::EnergyKernel<1024> << <ceil(s / (double)1024), 1024>> > (
		EnergyAtomic.cuda_arr,
		curr_x.cuda_arr,
		Const_Ind.size,
		Const_Ind.cuda_arr,
		Const_Pos.cuda_arr,
		startX, startY, startZ);
}
		
void Cuda_FixChosenConstraints::gradient(Cuda::Array<double>& X)
{
	Utils_Cuda_FixChosenConstraints::setZeroKernel << <grad.size, 1, 0, stream_gradient >> > (grad.cuda_arr);
	Utils_Cuda_FixChosenConstraints::gradientKernel << <Const_Ind.size, 3, 0, stream_gradient >> > (
		grad.cuda_arr,
		X.cuda_arr,
		startX, startY, startZ,
		Const_Ind.cuda_arr,
		Const_Pos.cuda_arr,
		Const_Ind.size);
}

Cuda_FixChosenConstraints::Cuda_FixChosenConstraints(const unsigned int numF, const unsigned int numV)
{
	hipStreamCreate(&stream_value);
	hipStreamCreate(&stream_gradient);
	Cuda::initIndices(mesh_indices, numF, numV, 0);
	Cuda::AllocateMemory(grad, (3 * numV) + (10 * numF));
	Cuda::AllocateMemory(EnergyAtomic, 1);
	Cuda::AllocateMemory(Const_Ind, 0);
	Cuda::AllocateMemory(Const_Pos, 0);
	//Choose the kind of constraints
	startX = mesh_indices.startVx;
	startY = mesh_indices.startVy;
	startZ = mesh_indices.startVz;
	
	//init host buffers...
	for (int i = 0; i < grad.size; i++) {
		grad.host_arr[i] = 0;
	}
	// Copy input vectors from host memory to GPU buffers.
	Cuda::MemCpyHostToDevice(grad);
}

Cuda_FixChosenConstraints::~Cuda_FixChosenConstraints() {
	hipStreamDestroy(stream_value);
	hipStreamDestroy(stream_gradient);
	hipGetErrorString(hipGetLastError());
	FreeMemory(grad);
	FreeMemory(EnergyAtomic);
	FreeMemory(Const_Ind);
	FreeMemory(Const_Pos);
}
