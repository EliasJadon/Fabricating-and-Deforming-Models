#include "hip/hip_runtime.h"
#include "Cuda_Grouping.cuh"
#include "Cuda_Minimizer.cuh"

namespace Utils_Cuda_Grouping {
	__device__ double3 sub(const double3 a, const double3 b)
	{
		return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
	}
	__device__ double dot(const double3 a, const double3 b)
	{
		return a.x * b.x + a.y * b.y + a.z * b.z;
	}
	__device__ double squared_norm(const double3 a)
	{
		return dot(a, a);
	}
	template<typename T>
	__global__ void setZeroKernel(T* vec)
	{
		vec[blockIdx.x] = 0;
	}
	template <unsigned int blockSize, typename T>
	__device__ void warpReduce(volatile T* sdata, unsigned int tid) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	
	__device__ double atomicAdd(double* address, double val, int flag)
	{
		unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
		unsigned long long int old = *address_as_ull, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));
			// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
		} while (assumed != old);
		return __longlong_as_double(old);
	}
	template <unsigned int blockSize>
	__global__ void sumOfArray(double* g_idata, unsigned int n) {
		extern __shared__ double sdata[blockSize];
		unsigned int tid = threadIdx.x;
		unsigned int i = blockIdx.x * (blockSize * 2) + tid;
		unsigned int gridSize = blockSize * 2 * gridDim.x;
		sdata[tid] = 0;
		while (i < n) { sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize; }
		__syncthreads();

		if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
		if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
		if (tid < 32) warpReduce<blockSize, double>(sdata, tid);
		if (tid == 0) atomicAdd(g_idata, sdata[0], 0);
	}

	__global__ void valueKernel(
		double* resAtomic,
		const double* curr_x,
		const unsigned int startX,
		const unsigned int startY,
		const unsigned int startZ,
		const int* Group_Ind,
		const unsigned int num_clusters,
		const unsigned int max_face_per_cluster)
	{
		int f1 = blockIdx.x;
		int f2 = blockIdx.y;
		int ci = blockIdx.z;
		int globalIndex = f1 + f2 * max_face_per_cluster + ci * max_face_per_cluster* max_face_per_cluster;
		if ((f1 > f2) &&
			(ci < num_clusters) &&
			(f1 < max_face_per_cluster) &&
			(f2 < max_face_per_cluster))
		{
			const unsigned int indexF1 = Group_Ind[ci * max_face_per_cluster + f1];
			const unsigned int indexF2 = Group_Ind[ci * max_face_per_cluster + f2];
			if (indexF1 != -1 && indexF2 != -1) {
				double3 NormalPos1 = make_double3(
					curr_x[indexF1 + startX],	//X-coordinate
					curr_x[indexF1 + startY],	//Y-coordinate
					curr_x[indexF1 + startZ]		//Z-coordinate
				);
				double3 NormalPos2 = make_double3(
					curr_x[indexF2 + startX],	//X-coordinate
					curr_x[indexF2 + startY],	//Y-coordinate
					curr_x[indexF2 + startZ]		//Z-coordinate
				);
				resAtomic[globalIndex] = Utils_Cuda_Grouping::squared_norm(Utils_Cuda_Grouping::sub(NormalPos1, NormalPos2));
			}
			else {
				resAtomic[globalIndex] = 0;
			}
		}
		else {
			resAtomic[globalIndex] = 0;
		}
	}

	__global__ void gradientKernel(
		double* grad,
		const double* X,
		const unsigned int startX,
		const unsigned int startY,
		const unsigned int startZ,
		const int* Group_Ind,
		const unsigned int num_clusters,
		const unsigned int max_face_per_cluster)
	{
		int f1 = blockIdx.x;
		int f2 = blockIdx.y;
		int ci = blockIdx.z;
		
		if ((f1 > f2) &&
			(ci < num_clusters) &&
			(f1 < max_face_per_cluster) &&
			(f2 < max_face_per_cluster)) 
		{
			const unsigned int indexF1 = Group_Ind[ci * max_face_per_cluster + f1];
			const unsigned int indexF2 = Group_Ind[ci * max_face_per_cluster + f2];
			if (indexF1 != -1 && indexF2 != -1) {
				if (threadIdx.x == 0)
					atomicAdd(&grad[indexF1 + startX], 2 * (X[indexF1 + startX] - X[indexF2 + startX]), 0);
				if (threadIdx.x == 1)
					atomicAdd(&grad[indexF2 + startX], -2 * (X[indexF1 + startX] - X[indexF2 + startX]), 0);
				if (threadIdx.x == 2)
					atomicAdd(&grad[indexF1 + startY], 2 * (X[indexF1 + startY] - X[indexF2 + startY]), 0);
				if (threadIdx.x == 3)
					atomicAdd(&grad[indexF2 + startY], -2 * (X[indexF1 + startY] - X[indexF2 + startY]), 0);
				if (threadIdx.x == 4)
					atomicAdd(&grad[indexF1 + startZ], 2 * (X[indexF1 + startZ] - X[indexF2 + startZ]), 0);
				if (threadIdx.x == 5)
					atomicAdd(&grad[indexF2 + startZ], -2 * (X[indexF1 + startZ] - X[indexF2 + startZ]), 0);
			}
		}
	}
}


double Cuda_Grouping::value(Cuda::Array<double>& curr_x) 
{
	/*Utils_Cuda_Grouping::setZeroKernel << <EnergyAtomic.size, 1 >> > (EnergyAtomic.cuda_arr);
	Cuda::CheckErr(hipDeviceSynchronize());
	Utils_Cuda_Grouping::valueKernel
		<< <dim3(max_face_per_cluster, max_face_per_cluster, num_clusters), 1 >> > (
			EnergyAtomic.cuda_arr,
			curr_x.cuda_arr,
			startX,
			startY,
			startZ,
			Group_Ind.cuda_arr,
			num_clusters,
			max_face_per_cluster);
	Cuda::CheckErr(hipDeviceSynchronize());
	
	Utils_Cuda_Grouping::sumOfArray<1024> << <ceil(EnergyAtomic.size / (double)1024), 1024 >> >
		(EnergyAtomic.cuda_arr, EnergyAtomic.size);

	Cuda::CheckErr(hipDeviceSynchronize());

	MemCpyDeviceToHost(EnergyAtomic,1);

	return EnergyAtomic.host_arr[0];*/
	return 0;
}
		
Cuda::Array<double>* Cuda_Grouping::gradient(Cuda::Array<double>& X)
{
	Utils_Cuda_Grouping::setZeroKernel << <grad.size, 1 >> > (grad.cuda_arr);
	Cuda::CheckErr(hipDeviceSynchronize());
	Utils_Cuda_Grouping::gradientKernel
		<< <dim3(max_face_per_cluster, max_face_per_cluster, num_clusters), 6 >> > (
			grad.cuda_arr,
			X.cuda_arr,
			startX,
			startY,
			startZ,
			Group_Ind.cuda_arr,
			num_clusters,
			max_face_per_cluster
			);
	Cuda::CheckErr(hipDeviceSynchronize());
	return &grad;
}

Cuda_Grouping::Cuda_Grouping(
	const unsigned int numF,
	const unsigned int numV,
	const ConstraintsType const_Type)
{
	Cuda::initIndices(mesh_indices, numF, numV, 0);
	Cuda::AllocateMemory(grad, (3 * numV) + (7 * numF));
	Cuda::AllocateMemory(EnergyAtomic, 1);
	Cuda::AllocateMemory(Group_Ind, 0);
	//Choose the kind of constraints
	if (const_Type == ConstraintsType::VERTICES) { 
		std::cout << "Cuda_Grouping class Error! Invalid ConstraintsType.\n";
		exit(1);
	}
	if (const_Type == ConstraintsType::NORMALS) { 
		startX = mesh_indices.startNx;
		startY = mesh_indices.startNy;
		startZ = mesh_indices.startNz;
	}
	if (const_Type == ConstraintsType::SPHERES) { 
		startX = mesh_indices.startCx;
		startY = mesh_indices.startCy;
		startZ = mesh_indices.startCz;
	}
	
	//init host buffers...
	for (int i = 0; i < grad.size; i++) {
		grad.host_arr[i] = 0;
	}
	// Copy input vectors from host memory to GPU buffers.
	Cuda::MemCpyHostToDevice(grad);
}

Cuda_Grouping::~Cuda_Grouping() {
	hipGetErrorString(hipGetLastError());
	FreeMemory(grad);
	FreeMemory(EnergyAtomic);
	FreeMemory(Group_Ind);
}
