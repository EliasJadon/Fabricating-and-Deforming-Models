#include "hip/hip_runtime.h"
#include "Cuda_Grouping.cuh"
#include "Cuda_Minimizer.cuh"

namespace Utils_Cuda_Grouping {
	__device__ double3 sub(const double3 a, const double3 b)
	{
		return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
	}
	__device__ double dot(const double3 a, const double3 b)
	{
		return a.x * b.x + a.y * b.y + a.z * b.z;
	}
	__device__ double squared_norm(const double3 a)
	{
		return dot(a, a);
	}
	template<typename T>
	__global__ void setZeroKernel(T* vec)
	{
		vec[blockIdx.x] = 0;
	}
	template <unsigned int blockSize, typename T>
	__device__ void warpReduce(volatile T* sdata, unsigned int tid) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	__device__ double atomicAdd(double* address, double val, int flag)
	{
		unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
		unsigned long long int old = *address_as_ull, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));
			// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
		} while (assumed != old);
		return __longlong_as_double(old);
	}

	

	__global__ void gradientKernel(
		double* grad,
		const double* X,
		const unsigned int startX,
		const unsigned int startY,
		const unsigned int startZ,
		const int* Group_Ind,
		const double3* Const_Pos,
		const unsigned int size)
	{
		int i = blockIdx.x;
		if (i < size) {
			if (threadIdx.x == 0)
				grad[Group_Ind[i] + startX] = 2 * (X[Group_Ind[i] + startX] - Const_Pos[i].x);
			if (threadIdx.x == 1)
				grad[Group_Ind[i] + startY] = 2 * (X[Group_Ind[i] + startY] - Const_Pos[i].y);
			if (threadIdx.x == 2)
				grad[Group_Ind[i] + startZ] = 2 * (X[Group_Ind[i] + startZ] - Const_Pos[i].z);
		}
	}
}


double Cuda_Grouping::value(Cuda::Array<double>& curr_x) 
{
	return 18;
}
		
void Cuda_Grouping::gradient(Cuda::Array<double>& X)
{
}

Cuda_Grouping::Cuda_Grouping(
	const unsigned int numF,
	const unsigned int numV,
	const ConstraintsType const_Type)
{
	Cuda::initIndices(mesh_indices, numF, numV, 0);
	Cuda::AllocateMemory(grad, (3 * numV) + (7 * numF));
	Cuda::AllocateMemory(EnergyAtomic, 1);
	Cuda::AllocateMemory(Group_Ind, 0);
	//Choose the kind of constraints
	if (const_Type == ConstraintsType::VERTICES) { 
		std::cout << "Cuda_Grouping class Error! Invalid ConstraintsType.\n";
		exit(1);
	}
	if (const_Type == ConstraintsType::NORMALS) { 
		startX = mesh_indices.startNx;
		startY = mesh_indices.startNy;
		startZ = mesh_indices.startNz;
	}
	if (const_Type == ConstraintsType::SPHERES) { 
		startX = mesh_indices.startCx;
		startY = mesh_indices.startCy;
		startZ = mesh_indices.startCz;
	}
	
	//init host buffers...
	for (int i = 0; i < grad.size; i++) {
		grad.host_arr[i] = 0;
	}
	// Copy input vectors from host memory to GPU buffers.
	Cuda::MemCpyHostToDevice(grad);
}

Cuda_Grouping::~Cuda_Grouping() {
	hipGetErrorString(hipGetLastError());
	FreeMemory(grad);
	FreeMemory(EnergyAtomic);
	FreeMemory(Group_Ind);
}
