#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <atomic>
#include <vector>
#include <mutex>
#include <iostream>
#include "CudaBasics.cuh"


namespace Cuda {
	hinge newHinge(int f0, int f1) {
		hinge a;
		a.f0 = f0;
		a.f1 = f1;
		return a;
	}
	
	void initCuda() {
		hipError_t cudaStatus;
		check_devices_properties();
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			hipGetErrorString(cudaStatus);
		}
		else {
			printf("hipSetDevice successfully!\n");
		}
	}

	void StopCudaDevice() {
		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		hipError_t cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			hipGetErrorString(cudaStatus);
		}
		else {
			printf("hipDeviceReset successfully!\n");
		}
	}

	void check_devices_properties() {
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipError_t cudaStatus;
		for (int i = 0; i < nDevices; i++) {
			hipDeviceProp_t prop;
			cudaStatus = hipGetDeviceProperties(&prop, i);
			hipGetErrorString(cudaStatus);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
			printf("  prop.maxThreadsPerBlock = %d\n", prop.maxThreadsPerBlock);
			printf("  prop.maxThreadsDim[0] = %d\n", prop.maxThreadsDim[0]);
			printf("  prop.maxThreadsDim[1] = %d\n", prop.maxThreadsDim[1]);
			printf("  prop.maxThreadsDim[2] = %d\n", prop.maxThreadsDim[2]);
			printf("  prop.maxGridSize[0] = %d\n", prop.maxGridSize[0]);
			printf("  prop.maxGridSize[1] = %d\n", prop.maxGridSize[1]);
			printf("  prop.maxGridSize[2] = %d\n", prop.maxGridSize[2]);
		}
	}
}

