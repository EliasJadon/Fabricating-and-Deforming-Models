#include "hip/hip_runtime.h"
#include "Cuda_SymmetricDirichlet.cuh"
#include "Cuda_Minimizer.cuh"

namespace Cuda {
	namespace SymmetricDirichlet {
		Array<double> grad, EnergyAtomic, EnergyVec, restShapeArea;
		Array<double3> D1d, D2d;
		Array<int3> restShapeF;
		unsigned int num_faces, num_vertices;

		
		
		__device__ double3 add(double3 a, double3 b)
		{
			return make_double3(a.x + b.x, a.y + b.y, a.z + b.z);
		}
		__device__ double3 sub(const double3 a, const double3 b)
		{
			return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
		}
		__device__ double dot(const double3 a, const double3 b)
		{
			return a.x * b.x + a.y * b.y + a.z * b.z;
		}
		__device__ double3 mul(const double a, const double3 b)
		{
			return make_double3(a * b.x, a * b.y, a * b.z);
		}
		__device__ double squared_norm(const double3 a)
		{
			return dot(a, a);
		}
		__device__ double norm(const double3 a)
		{
			return sqrt(squared_norm(a));
		}
		__device__ double3 normalize(const double3 a)
		{
			return mul(1.0f / norm(a), a);
		}
		__device__ double3 cross(const double3 a, const double3 b)
		{
			return make_double3(
				a.y * b.z - a.z * b.y,
				a.z * b.x - a.x * b.z,
				a.x * b.y - a.y * b.x
			);
		}

		template<unsigned int row1, unsigned int col1, unsigned int row2, unsigned int col2>
		__device__ void mulMatrix(const double res[row1][col2], const double a[row1][col1], const double b[row2][col2]) {
			// Multiplying matrix a and b and storing in array mult.
			for (int i = 0; i < row1; ++i)
				for (int j = 0; j < col2; ++j)
					for (int k = 0; k < col1; ++k)
					{
						res[i][j] += a[i][k] * b[k][j];
					}
		}

		__device__ double atomicAdd(double* address, double val, int flag)
		{
			unsigned long long int* address_as_ull =
				(unsigned long long int*)address;
			unsigned long long int old = *address_as_ull, assumed;

			do {
				assumed = old;
				old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
						__longlong_as_double(assumed)));

				// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
			} while (assumed != old);

			return __longlong_as_double(old);
		}

		template <unsigned int blockSize, typename T>
		__device__ void warpReduce(volatile T* sdata, unsigned int tid) {
			if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
			if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
			if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
			if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
			if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
			if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
		}








		
		
		

		
		__device__ void local_basis(
			const double3 CurrV0,	
			const double3 CurrV1,	
			const double3 CurrV2,	
			double3& B1,		
			double3& B2)
		{
			double3 v1 = normalize(sub(CurrV1, CurrV0));
			double3 t = sub(CurrV2, CurrV0);
			double3 v3 = normalize(cross(v1, t));
			double3 v2 = normalize(cross(v1, v3));
			B1 = v1;
			B2 = mul(-1, v2);
		}
			   		 	  
		template<unsigned int blockSize>
		__global__ void EnergyKernel(
			double* resAtomic,
			double* Energy,
			const double* X,
			const int3* restShapeF,
			const double* restShapeArea,
			const double3* D1d,
			const double3* D2d,
			const unsigned int num_faces,
			const unsigned int num_vertices)
		{
			extern __shared__ double energy_value[blockSize];
			unsigned int tid = threadIdx.x;
			unsigned int face_index = blockIdx.x * blockSize + tid;
			*resAtomic = 0;
			__syncthreads();

			if (face_index < num_faces) {
				unsigned int x0 = restShapeF[face_index].x;
				unsigned int x1 = restShapeF[face_index].y;
				unsigned int x2 = restShapeF[face_index].z;
				double3 CurrV0 = make_double3(X[x0], X[x0 + num_vertices], X[x0 + 2 * num_vertices]);
				double3 CurrV1 = make_double3(X[x1], X[x1 + num_vertices], X[x1 + 2 * num_vertices]);
				double3 CurrV2 = make_double3(X[x2], X[x2 + num_vertices], X[x2 + 2 * num_vertices]);
				double3 Dx = D1d[face_index];
				double3 Dy = D2d[face_index];
				double area = restShapeArea[face_index];

				double3 B1, B2;
				local_basis(CurrV0, CurrV1, CurrV2, B1, B2);

				double3 Xi = make_double3(
					dot(CurrV0, B1),
					dot(CurrV1, B1),
					dot(CurrV2, B1)
				);
				double3 Yi = make_double3(
					dot(CurrV0, B2),
					dot(CurrV1, B2),
					dot(CurrV2, B2)
				);
				//prepare jacobian		
				double a = dot(Dx, Xi);
				double b = dot(Dx, Yi);
				double c = dot(Dy, Xi);
				double d = dot(Dy, Yi);
				double det = a * d - b * c;

				//till now is updateX
				double det2 = det * det;
				double a2 = a * a;
				double b2 = b * b;
				double c2 = c * c;
				double d2 = d * d;
				double Fnorm = a2 + b2 + c2 + d2;
				//now, value...
				double val = 0.5 * area * (1 + 1 / det2) * Fnorm;
				Energy[face_index] = val;
				energy_value[tid] = val;
			}
			else {
				energy_value[tid] = 0;
			}

			__syncthreads();

			if (blockSize >= 1024) { if (tid < 512) { energy_value[tid] += energy_value[tid + 512]; } __syncthreads(); }
			if (blockSize >= 512) { if (tid < 256) { energy_value[tid] += energy_value[tid + 256]; } __syncthreads(); }
			if (blockSize >= 256) { if (tid < 128) { energy_value[tid] += energy_value[tid + 128]; } __syncthreads(); }
			if (blockSize >= 128) { if (tid < 64) { energy_value[tid] += energy_value[tid + 64]; } __syncthreads(); }
			if (tid < 32) warpReduce<blockSize, double>(energy_value, tid);
			if (tid == 0) atomicAdd(resAtomic, energy_value[0], 0);
		}

		double value() {
			EnergyKernel<1024> << <ceil(num_faces / (double)1024), 1024 >> > (
				EnergyAtomic.cuda_arr,
				EnergyVec.cuda_arr,
				Cuda::Minimizer::curr_x.cuda_arr,
				restShapeF.cuda_arr,
				restShapeArea.cuda_arr,
				D1d.cuda_arr,
				D2d.cuda_arr,
				num_faces,
				num_vertices);
			CheckErr(hipDeviceSynchronize());
			MemCpyDeviceToHost(EnergyAtomic);
			return EnergyAtomic.host_arr[0];
		}






		//__device__ void gradient(double* grad)
		//{

		//	double4 de_dJ = make_double4(
		//		area * (a + a / det2 - d * Fnorm / det3),
		//		area * (b + b / det2 + c * Fnorm / det3),
		//		area * (c + c / det2 + b * Fnorm / det3),
		//		area * (d + d / det2 - a * Fnorm / det3)
		//	);

		//	double dE_dX[9];
		//	dE_dX = de_dJ/*1,4*/ * dJ_dX(fi)/*4,9*/;
		//		
		//	grad[x0]						+= dE_dX[0];
		//	grad[x0 + num_vertices]			+= dE_dX[3];
		//	grad[x0 + (2 * num_vertices)]	+= dE_dX[6];

		//	grad[x1]						+= dE_dX[1];
		//	grad[x1 + num_vertices]			+= dE_dX[4];
		//	grad[x1 + (2 * num_vertices)]	+= dE_dX[7];

		//	grad[x2]						+= dE_dX[2];
		//	grad[x2 + num_vertices]			+= dE_dX[5];
		//	grad[x2 + (2 * num_vertices)]	+= dE_dX[8];
		//}

		//__device__ double**/*[3][9]*/ dB1_dX(const double3 vec10) 
		//{
		//	double Norm = norm(vec10);
		//	double Norm3 = Norm * Norm * Norm;

		//	double dB1x_dx0 = -(pow(vec10.y, 2) + pow(vec10.z, 2)) / Norm3;
		//	double dB1y_dy0 = -(pow(vec10.x, 2) + pow(vec10.z, 2)) / Norm3;
		//	double dB1z_dz0 = -(pow(vec10.x, 2) + pow(vec10.y, 2)) / Norm3;
		//	double dB1x_dy0 = (vec10.y * vec10.x) / Norm3;
		//	double dB1x_dz0 = (vec10.z * vec10.x) / Norm3;
		//	double dB1y_dz0 = (vec10.z * vec10.y) / Norm3;
		//	double g[3][9];
		//	
		//	g[0][0] = dB1x_dx0; 
		//	g[0][1] = -dB1x_dx0; 
		//	g[0][2] = 0; 
		//	g[0][3] = dB1x_dy0;
		//	g[0][4] = -dB1x_dy0;
		//	g[0][5] = 0;
		//	g[0][6] = dB1x_dz0;
		//	g[0][7] = -dB1x_dz0;
		//	g[0][8] = 0;
		//			
		//	g[1][0] = dB1x_dy0;
		//	g[1][1] = -dB1x_dy0;
		//	g[1][2] = 0;
		//	g[1][3] = dB1y_dy0;
		//	g[1][4] = -dB1y_dy0; 
		//	g[1][5] = 0;
		//	g[1][6] = dB1y_dz0; 
		//	g[1][7] = -dB1y_dz0;
		//	g[1][8] = 0;
		//		
		//	g[2][0] = dB1x_dz0;
		//	g[2][1] = -dB1x_dz0;
		//	g[2][2] = 0;
		//	g[2][3] = dB1y_dz0;
		//	g[2][4] = -dB1y_dz0;
		//	g[2][5] = 0;
		//	g[2][6] = dB1z_dz0;
		//	g[2][7] = -dB1z_dz0;
		//	g[2][8] = 0;
		//	return g;
		//}

		//__device__ double**/*[3][9]*/ dB2_dX(const double3 vec10,const double3 vec20) 
		//{
		//	double3 b2 = mul(-1,(cross(vec10,cross(vec10,vec20))));
		//	double NormB2 = norm(b2);
		//	double NormB2_2 = pow(NormB2, 2);

		//	double3 dxyz[6];
		//	dxyz[0] = make_double3(
		//		-vec10.y * vec20.y - vec10.z * vec20.z,
		//		2 * vec10.x * vec20.y - vec10.y * vec20.x,
		//		-vec10.z * vec20.x + 2 * vec10.x * vec20.z
		//	);
		//	dxyz[1] = make_double3(
		//		-vec10.x * vec20.y + 2 * vec10.y * vec20.x,
		//		-vec10.z * vec20.z - vec20.x * vec10.x,
		//		2 * vec10.y * vec20.z - vec10.z * vec20.y
		//	);
		//	dxyz[2] = make_double3(
		//		2 * vec10.z * vec20.x - vec10.x * vec20.z,
		//		-vec10.y * vec20.z + 2 * vec10.z * vec20.y,
		//		-vec10.x * vec20.x - vec10.y * vec20.y
		//	);
		//	dxyz[3] = make_double3(
		//		pow(vec10.y, 2) + pow(vec10.z, 2),
		//		-vec10.x * vec10.y,
		//		-vec10.x * vec10.z
		//	);
		//	dxyz[4] = make_double3(
		//		-vec10.y * vec10.x,
		//		pow(vec10.z, 2) + pow(vec10.x, 2),
		//		-vec10.z * vec10.y
		//	);
		//	dxyz[5] = make_double3(
		//		-vec10.x * vec10.z,
		//		-vec10.z * vec10.y,
		//		pow(vec10.x, 2) + pow(vec10.y, 2)
		//	);

		//	double dnorm[6];
		//	dnorm[0] = dot(b2, dxyz[0]) / NormB2;
		//	dnorm[1] = dot(b2, dxyz[1]) / NormB2;
		//	dnorm[2] = dot(b2, dxyz[2]) / NormB2;
		//	dnorm[3] = dot(b2, dxyz[3]) / NormB2;
		//	dnorm[4] = dot(b2, dxyz[4]) / NormB2;
		//	dnorm[5] = dot(b2, dxyz[5]) / NormB2;

		//	double g[3][9];
		//	g[0][1] = (dxyz[0].x * NormB2 - b2.x * dnorm[0]) / NormB2_2;
		//	g[0][2] = (dxyz[3].x * NormB2 - b2.x * dnorm[3]) / NormB2_2;
		//	g[0][0] = -g[0][1] - g[0][2];
		//	g[0][4] = (dxyz[1].x * NormB2 - b2.x * dnorm[1]) / NormB2_2;
		//	g[0][5] = (dxyz[4].x * NormB2 - b2.x * dnorm[4]) / NormB2_2;
		//	g[0][3] = -g[0][4] - g[0][5];
		//	g[0][7] = (dxyz[2].x * NormB2 - b2.x * dnorm[2]) / NormB2_2;
		//	g[0][8] = (dxyz[5].x * NormB2 - b2.x * dnorm[5]) / NormB2_2;
		//	g[0][6] = -g[0][7] - g[0][8];
		//	
		//	g[1][1] = (dxyz[0].y * NormB2 - b2.y * dnorm[0]) / NormB2_2;
		//	g[1][2] = (dxyz[3].y * NormB2 - b2.y * dnorm[3]) / NormB2_2;
		//	g[1][0] = -g[1][1] - g[1][2];
		//	g[1][4] = (dxyz[1].y * NormB2 - b2.y * dnorm[1]) / NormB2_2;
		//	g[1][5] = (dxyz[4].y * NormB2 - b2.y * dnorm[4]) / NormB2_2;
		//	g[1][3] = -g[1][4] - g[1][5];
		//	g[1][7] = (dxyz[2].y * NormB2 - b2.y * dnorm[2]) / NormB2_2;
		//	g[1][8] = (dxyz[5].y * NormB2 - b2.y * dnorm[5]) / NormB2_2;
		//	g[1][6] = -g[1][7] - g[1][8];

		//	g[2][1] = (dxyz[0].z * NormB2 - b2.z * dnorm[0]) / NormB2_2;
		//	g[2][2] = (dxyz[3].z * NormB2 - b2.z * dnorm[3]) / NormB2_2;
		//	g[2][0] = -g[2][1] - g[2][2];
		//	g[2][4] = (dxyz[1].z * NormB2 - b2.z * dnorm[1]) / NormB2_2;
		//	g[2][5] = (dxyz[4].z * NormB2 - b2.z * dnorm[4]) / NormB2_2;
		//	g[2][3] = -g[2][4] - g[2][5];
		//	g[2][7] = (dxyz[2].z * NormB2 - b2.z * dnorm[2]) / NormB2_2;
		//	g[2][8] = (dxyz[5].z * NormB2 - b2.z * dnorm[5]) / NormB2_2;
		//	g[2][6] = -g[2][7] - g[2][8];
		//	
		//	return g;
		//}

		//__device__ double** /*4, 9*/ dJ_dX(
		//	const double3 Dx,
		//	const double3 Dy,
		//	const double3 B1,
		//	const double3 B2,
		//	const double3 CurrV0,
		//	const double3 CurrV1,
		//	const double3 CurrV2)
		//{
		//	double dV0_dX[3][9] = { 0 }, dV1_dX[3][9] = { 0 }, dV2_dX[3][9] = { 0 };
		//	dV0_dX[0][0] = 1; dV0_dX[1][3] = 1; dV0_dX[2][6] = 1;
		//	dV1_dX[0][1] = 1; dV1_dX[1][4] = 1; dV1_dX[2][7] = 1;
		//	dV2_dX[0][2] = 1; dV2_dX[1][5] = 1; dV2_dX[2][8] = 1;

		//	double YY[3][9], XX[3][9];
		//	double db1_dX[3][9] = dB1_dX(vec10);
		//	double db2_dX[3][9] = dB2_dX(vec10, vec20);
		//	XX <<
		//		(CurrV0 * db1_dX + B1 * dV0_dX),
		//		(CurrV1 * db1_dX + B1 * dV1_dX),
		//		(CurrV2 * db1_dX + B1 * dV2_dX);
		//	YY <<
		//		(CurrV0 * db2_dX + B2 * dV0_dX),
		//		(CurrV1 * db2_dX + B2 * dV1_dX),
		//		(CurrV2 * db2_dX + B2 * dV2_dX);

		//	double dJ[4][9];
		//	dJ[0] = Dx * XX;
		//	dJ[1] = Dx * YY;
		//	dJ[2] = Dy * XX;
		//	dJ[3] = Dy * YY;
		//	return dJ;
		//}

		
		void FreeAllVariables() {
			hipGetErrorString(hipGetLastError());
			FreeMemory(restShapeF);
			FreeMemory(D1d);
			FreeMemory(D2d);
			FreeMemory(EnergyVec);
			FreeMemory(restShapeArea);
			FreeMemory(grad);
			FreeMemory(EnergyAtomic);
		}
	}
}
